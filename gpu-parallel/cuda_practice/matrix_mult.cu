#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void matrix_mult(float* a, float* b, float* c, int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;


    if (row < N && col < N) {
         float temp = 0;
         for (int i = 0; i < N; ++i)
        {
            temp += d_a[row*width+i]*d_b[i*width+col];
        }
         d_c[row*width+col] = temp;
    }
    __syncthreads();
}

int main()
{
    int N = 1000;
    float *a, *b, *c;

    a = (float*)malloc(sizeof(float)*N*N);
    b = (float*)malloc(sizeof(float)*N*N);
    c = (float*)malloc(sizeof(float)*N*N);

    float *d_a, *d_b, *d_c;

    d_a = hipMalloc((void**)&d_a, sizeof(float) * N*N);
    d_b = hipMalloc((void**)&d_b, sizeof(float) * N*N);
    d_c = hipMalloc((void**)&d_c, sizeof(float) * N*N);

    for(int i = 0; i < N*N; i++)
    {
      a[i] = 1.0;
      b[i] = 2.0;
    }

    hipMemcpy(d_a, a, size(float)*N*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size(float)*N*N, hipMemcpyHostToDevice);

    int blockSize = 32;
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimGrid(ceil(N/float(blockSize)),ceil(N/float(blockSize)),1);
    matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, size(float)*N*N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            printf("%f ", c[i*N+j]);
        }
        printf("\n");
    }
    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Deallocate host memory
    free(a);
    free(b);
    free(c);
    return 0;
}