#include "hip/hip_runtime.h"
#include"cg_gpu_parallel.h"
#include"LAS_gpu_parallel.h"
#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define CACHE_BLOCK_SIZE 64 // bytes
#define BLOCK_DIM 32
// compute step size


// check where to use 
extern "C" float* cg(unsigned int size, float* A, float* b, float* init_g, float epsilon, int debug)
{
	float*  solution;
	float*  residual;
	float*  residual_prev;
	float*  search_direction;
	float*  intermediate_comp;
	float alpha;
	float beta;
	float num;
	float denum;
	float err;
	int i = 0;
    printf("Size : %d \n", size);
	printf("Grid: %d , Block: %d\n", (size + BLOCK_DIM - 1) / BLOCK_DIM, BLOCK_DIM);	
	dim3 GridDim( (size + BLOCK_DIM - 1) / BLOCK_DIM ,1,1);
	dim3 BlockDim(BLOCK_DIM,1,1);


	solution = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	residual = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	residual_prev = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	search_direction = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	intermediate_comp = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);



	scalar_vector_mult_inplace(size,solution,0);
	scalar_vector_mult_inplace(size,residual,0);
	scalar_vector_mult_inplace(size,residual_prev,0);
	scalar_vector_mult_inplace(size,search_direction,0);
	scalar_vector_mult_inplace(size,intermediate_comp,0);

	vector_copy(size, solution, init_g);
	
	compute_residual_gpu(size, A, b, solution, residual,GridDim,BlockDim);
	hipDeviceSynchronize();
	vector_copy(size,search_direction,residual);
	
	scalar_vector_mult_inplace(size,search_direction,-1);
	
	err = dot_product(size, residual, residual);

	err = sqrt(err);

    	float* d_A, *d_sd, *d_ic;
    	//hipMalloc((void**)&d_A, sizeof(float)*size*size);
    	hipError_t erra;
	erra = hipMalloc((void**)&d_A, sizeof(float)*size*size);
	if (erra != hipSuccess) printf("hipMalloc d_A failed: %s\n", hipGetErrorString(erra));
	hipMalloc((void**)&d_sd, sizeof(float)*size);
    	hipMalloc((void**)&d_ic, sizeof(float)*size);
	hipMemcpy(d_A, A, sizeof(float)*size*size, hipMemcpyHostToDevice);

	while (err > epsilon)
	{
		
		printf("iteration %d\n",i);
        	hipMemcpy(d_sd, search_direction, sizeof(float)*size, hipMemcpyHostToDevice);
		hipMemcpy(d_ic, intermediate_comp, sizeof(float)*size, hipMemcpyHostToDevice);
		if (d_A == NULL || d_sd == NULL || d_ic == NULL) {
 		   printf("Device pointers not allocated correctly\n");
		}
		printf("Launching kernel with GridDim=%d, BlockDim=%d\n", GridDim, BlockDim);
		test_access<<<GridDim,BlockDim>>>(d_sd);
		hipDeviceSynchronize();
		hipError_t errs = hipPeekAtLastError();
		if (errs != hipSuccess)
    			printf("Pre-launch error: %s\n", hipGetErrorString(errs));		
        	matrix_vector_mult<<<GridDim,BlockDim>>>(size,d_A,d_sd,d_ic);
		
		hipDeviceSynchronize();
		hipError_t errc = hipGetLastError();
		if (errc != hipSuccess)
			printf("CUDA kernel error: %s\n", hipGetErrorString(errc));
        	hipMemcpy(intermediate_comp, d_ic, sizeof(float)*size,hipMemcpyDeviceToHost);
		
		num = dot_product(size, residual, residual);
		denum = dot_product(size, search_direction, intermediate_comp);
		
		alpha =  num / denum;
	
		vector_add(size,solution,search_direction,alpha,solution);
		
		//vector_copy(size,residual_prev,residual);
		
		vector_add(size,residual,intermediate_comp,alpha,residual);

		denum = num;
		
		num = dot_product(size, residual,residual);

		//denum = dot_product(size, residual_prev, residual_prev);

		beta = num / denum;

		err = sqrt(num);
		
		vector_copy(size,intermediate_comp,residual);
		
		scalar_vector_mult_inplace(size,intermediate_comp,-1);
		
		vector_add(size,intermediate_comp,search_direction,beta,search_direction);

		i++;
	}
	
	printf("Number of iterations: %d\nFinal epsilon: %.12lf\n", i, err);
	free(residual);
	free(residual_prev);
	free(search_direction);
	free(intermediate_comp);
    hipFree(d_A);
    hipFree(d_sd);
    hipFree(d_ic);
	return solution;
}	
