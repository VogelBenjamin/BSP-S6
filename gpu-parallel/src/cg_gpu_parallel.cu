#include "hip/hip_runtime.h"
#include"cg_gpu_parallel.h"
#include"LAS_gpu_parallel.h"
#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define CACHE_BLOCK_SIZE 64 // bytes
#define BLOCK_DIM 32
// compute step size


// check where to use 
float* cg(unsigned int size, float* A, float* b, float* init_g, float epsilon, int debug)
{
	float*  solution;
	float*  residual;
	float*  residual_prev;
	float*  search_direction;
	float*  intermediate_comp;
	float alpha;
	float beta;
	float num;
	float denum;
	float err;
	int i = 0;

	
	dim3 GridDim(ceil(size/BLOCK_DIM),ceil(size/BLOCK_DIM),1);
    dim3 BlockDim(BLOCK_DIM,BLOCK_DIM,1);


	solution = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	residual = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	residual_prev = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	search_direction = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	intermediate_comp = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);



	scalar_vector_mult_inplace(size,solution,0);
	scalar_vector_mult_inplace(size,residual,0);
	scalar_vector_mult_inplace(size,residual_prev,0);
	scalar_vector_mult_inplace(size,search_direction,0);
	scalar_vector_mult_inplace(size,intermediate_comp,0);

	vector_copy(size, solution, init_g);
	
	compute_residual_gpu(size, A, b, solution, residual,GridDim,BlockDim);
	
	vector_copy(size,search_direction,residual);
	
	scalar_vector_mult_inplace(size,search_direction,-1);
	
	err = dot_product(size, residual, residual);

	err = sqrt(err);

    float* d_A, *d_sd, *d_ic;
    hipMalloc((void**)&d_A, sizeof(float)*size*size);
    hipMalloc((void**)&d_sd, sizeof(float)*size);
    hipMalloc((void**)&d_ic, sizeof(float)*size);

	while (err > epsilon)
	{
		
	hipMemcpy(d_A, A, sizeof(float)*size*size, hipMemcpyHostToDevice);
        hipMemcpy(d_sd, search_direction, sizeof(float)*size, hipMemcpyHostToDevice);
		
        matrix_vector_mult<<<GridDim,BlockDim>>>(size,A,search_direction,intermediate_comp);
		
        hipMemcpy(intermediate_comp, d_ic, sizeof(float)*size,hipMemcpyDeviceToHost);

		num = dot_product(size, residual, residual);

		denum = dot_product(size, search_direction, intermediate_comp);
		
		alpha =  num / denum;
	
		vector_add(size,solution,search_direction,alpha,solution);
		
		//vector_copy(size,residual_prev,residual);
		
		vector_add(size,residual,intermediate_comp,alpha,residual);

		denum = num;
		
		num = dot_product(size, residual,residual);

		//denum = dot_product(size, residual_prev, residual_prev);

		beta = num / denum;

		err = sqrt(num);
		
		vector_copy(size,intermediate_comp,residual);
		
		scalar_vector_mult_inplace(size,intermediate_comp,-1);
		
		vector_add(size,intermediate_comp,search_direction,beta,search_direction);

		i++;
	}
	
	printf("Number of iterations: %d\nFinal epsilon: %.12lf\n", i, err);
	free(residual);
	free(residual_prev);
	free(search_direction);
	free(intermediate_comp);
    hipFree(d_A);
    hipFree(d_sd);
    hipFree(d_ic);
	return solution;
}	
