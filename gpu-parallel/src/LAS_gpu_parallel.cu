#include "hip/hip_runtime.h"
#include"LAS_gpu_parallel.h"
#include<hip/hip_runtime.h>

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

// assume 1d Grid and 1d Block
__global__ void matrix_vector_mult(unsigned int size, float* matrix, float* vector, float* vector_storage)
{
	int row = blockDim.x*blockIdx.x + threadIdx.x;
  //int col = blockDim.y*blockIdx.y + threadIdx.y

  if (row < size)
  {
      float acc = 0;
      for (int i = 0; i < size; ++i)
      {
          acc += matrix[row*size+i]*vector[row];
      }
      vector_storage[row] = acc;
  }
	__syncthreads();
	return;
}

float dot_product(unsigned int size, float* vector_1, float* vector_2)
{
	float storage = 0;
	for (unsigned int i = 0; i < size; ++i)
	{
		storage += vector_1[i]*vector_2[i]; 
	}
	return storage;
}

void vector_add(unsigned int size, float* vector_1, float* vector_2, float alpha, float* vector_storage)
{
	for (unsigned int i = 0; i < size; ++i)
	{
		vector_storage[i] = vector_1[i] + alpha*vector_2[i];	
	}
	return;
}

void vector_sub(unsigned int size, float* vector_1, float* vector_2, float alpha, float* vector_storage)
{
        for (unsigned int i = 0; i < size; ++i)
        {
                vector_storage[i] = vector_1[i] - alpha*vector_2[i];
        }
        return;
}

void vector_copy(unsigned int size, float* vector_out, float* vector_in)
{
	for (unsigned int i = 0; i < size; ++i)
        {
                vector_out[i] = vector_in[i];
        }
	return;
}

void scalar_vector_mult_inplace(unsigned int size,float* vector, float alpha)
{
	for (unsigned int i = 0; i < size; ++i)
	{
			vector[i] = alpha*vector[i];
	}
    return;
}

void compute_residual_gpu(unsigned int size, float* A, float* b, float* x, float* vector_storage, dim3 GridDim, dim3 BlockDim)
{
	matrix_vector_mult<<<GridDim,BlockDim>>>(size,A,x,vector_storage);
	vector_sub(size,vector_storage,b,1.0,vector_storage);
	return;
}

void print_vector(unsigned int size, float* vector)
{
	for (unsigned int i = 0; i < size; ++i)
	{
			printf("%lf ",vector[i]);
	}
	printf("\n");
}


/*
// assume 1d Grid and 1d Block
__global__ void dot_product(unsigned int size, float* vector_1, float* vector_2, float* vector_out)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < size)
  {
	  vector_out[i] = vector_1[i]*vector_2[i];
  }
  __syncthreads();
}

__global__ void vector_add(unsigned int size, float* vector_1, float* vector_2, float alpha, float* vector_storage)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < size)
  {
	  vector_out[i] = vector_1[i]+vector_2[i];
  }
  __syncthreads();
}

__global__ void vector_sub(unsigned int size, float* vector_1, float* vector_2, float alpha, float* vector_storage)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < size)
  {
	  vector_out[i] = vector_1[i]-vector_2[i];
  }
  __syncthreads();
}

__global__ void vector_copy(unsigned int size, float* vector_out, float* vector_in)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < size)
  {
	  vector_out[i] = vector_in[i];
  }
  __syncthreads();
}

__global__ void scalar_vector_mult_inplace(unsigned int size,float* vector, float alpha)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < size)
  {
	  vector[i] = alpha*vector[i];
  }
  __syncthreads();
}
*/
