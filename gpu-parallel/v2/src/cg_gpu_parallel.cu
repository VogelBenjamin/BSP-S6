#include "hip/hip_runtime.h"
#include"cg_gpu_parallel.h"
#include"LAS_gpu_parallel.h"
#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define CACHE_BLOCK_SIZE 64 // bytes
#define BLOCK_DIM 64
// compute step size


// check where to use 
float* cg(unsigned int size, float* A, float* b, float* init_g, float epsilon, int debug)
{
	float*  solution;
	float*  residual;
	float*  residual_prev;
	float*  search_direction;
	float*  intermediate_comp;
	float *num = (float*)malloc(sizeof(float));
	float *one = (float*)malloc(sizeof(float));
	float err;
	int i = 0;
	*one = 1.0f;
    printf("Size : %d \n", size);
	printf("Grid: %d , Block: %d\n", (size + BLOCK_DIM - 1) / BLOCK_DIM, BLOCK_DIM);	
	dim3 GridDim( (size + BLOCK_DIM - 1) / BLOCK_DIM ,1,1);
	dim3 BlockDim(BLOCK_DIM,1,1);

	// allocate host data
	solution = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	residual = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	residual_prev = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	search_direction = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	intermediate_comp = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);

	// allocate device data
	float* d_A, *d_b, *d_sd, *d_ic, *d_sol, *d_res, *d_resp;

	hipMalloc((void**)&d_A, sizeof(float)*size*size);
	hipMalloc((void**)&d_b, sizeof(float)*size);
	hipMalloc((void**)&d_sd, sizeof(float)*size);
	hipMalloc((void**)&d_ic, sizeof(float)*size);
	hipMalloc((void**)&d_sol, sizeof(float)*size);
	hipMalloc((void**)&d_res, sizeof(float)*size);
	hipMalloc((void**)&d_resp, sizeof(float)*size);

	float*d_num, *d_denum, *d_one;
	hipMalloc((void**)&d_num, sizeof(float));
	hipMalloc((void**)&d_denum, sizeof(float));
	hipMalloc((void**)&d_one, sizeof(float));

	hipMemcpy(d_one,one,sizeof(float),hipMemcpyHostToDevice);

	// initialise device data
	
	hipError_t errs_init;
	
	scalar_vector_mult_inplace<<<GridDim,BlockDim>>>(size,d_sol,0);
	
	scalar_vector_mult_inplace<<<GridDim,BlockDim>>>(size,d_res,0);
	
	scalar_vector_mult_inplace<<<GridDim,BlockDim>>>(size,d_resp,0);
	
	scalar_vector_mult_inplace<<<GridDim,BlockDim>>>(size,d_sd,0);
	
	scalar_vector_mult_inplace<<<GridDim,BlockDim>>>(size,d_ic,0);

	scalar_vector_mult_inplace<<<GridDim,BlockDim>>>(1,d_num,0);

	scalar_vector_mult_inplace<<<GridDim,BlockDim>>>(1,d_denum,0);
	
	hipMemcpy(d_A, A, sizeof(float)*size*size, hipMemcpyHostToDevice);
	
	hipMemcpy(d_sol, init_g, sizeof(float)*size, hipMemcpyHostToDevice);

	hipMemcpy(d_b, b, sizeof(float)*size, hipMemcpyHostToDevice);

	errs_init = hipPeekAtLastError();
	if (errs_init != hipSuccess)
		printf("Pre-launch error init 8: %s\n", hipGetErrorString(errs_init));
	
	
	// residual calc
	// A = d_A , d_sol = solution , d_ic = residual
	hipError_t errs_ferr;

	matrix_vector_mult<<<GridDim,BlockDim>>>(size,d_A,d_sol,d_res);

	hipDeviceSynchronize();

	vector_sub<<<GridDim,BlockDim>>>(size,d_res,d_b,d_one,d_one,d_res);

	hipDeviceSynchronize();

	vector_copy<<<GridDim,BlockDim>>>(size,d_sd,d_res);
	
	hipDeviceSynchronize();

	scalar_vector_mult_inplace<<<GridDim,BlockDim>>>(size,d_sd,-1);

	
	errs_ferr = hipPeekAtLastError();	
	if (errs_ferr != hipSuccess)
		printf("Pre-launch error fmvm 4: %s\n", hipGetErrorString(errs_ferr));

	hipDeviceSynchronize();

	dot_product<<<GridDim,BlockDim>>>(size, d_res, d_res, d_num);
	hipMemcpy(num,d_num,sizeof(float),hipMemcpyDeviceToHost);
	err = sqrt(*num);

	printf("Launching kernel with GridDim=%d, BlockDim=%d\n", GridDim, BlockDim);
	
	while (err > epsilon)
	{
		hipMemset(d_num, 0, sizeof(float));

		hipMemset(d_denum, 0, sizeof(float));
		
		//printf("Error: %f\n",err);	
		hipError_t errc;
		// compute alpha
        matrix_vector_mult<<<GridDim,BlockDim>>>(size,d_A,d_sd,d_ic);
		
		dot_product<<<GridDim,BlockDim>>>(size, d_res, d_res, d_num);

		hipDeviceSynchronize();
		
		dot_product<<<GridDim,BlockDim>>>(size, d_sd, d_ic, d_denum);
		errc = hipGetLastError();
		if (errc != hipSuccess)
			printf("CUDA kernel error: %s\n", hipGetErrorString(errc));
		hipDeviceSynchronize();

		// update residual and search direction
		vector_add<<<GridDim,BlockDim>>>(size,d_sol,d_sd,d_num,d_denum,d_sol);
		
		vector_add<<<GridDim,BlockDim>>>(size,d_res,d_ic,d_num,d_denum,d_res);
		errc = hipGetLastError();
		if (errc != hipSuccess)
			printf("CUDA kernel error: %s\n", hipGetErrorString(errc));

		hipDeviceSynchronize();

		// compute beta
		vector_copy<<<GridDim,BlockDim>>>(1,d_denum, d_num);
		
		hipMemset(d_num, 0, sizeof(float)); // already synchronizes device

		dot_product<<<GridDim,BlockDim>>>(size, d_res,d_res,d_num);

		hipMemcpy(num,d_num,sizeof(float),hipMemcpyDeviceToHost); // synchronizes
		err = sqrt(*num);
		
		vector_copy<<<GridDim,BlockDim>>>(size,d_ic,d_res);
		
		hipDeviceSynchronize();

		scalar_vector_mult_inplace<<<GridDim,BlockDim>>>(size,d_ic,-1);

		hipDeviceSynchronize();

		vector_add<<<GridDim,BlockDim>>>(size,d_ic,d_sd,d_num,d_denum,d_sd);

		errc = hipGetLastError();
		if (errc != hipSuccess)
			printf("CUDA kernel error: %s\n", hipGetErrorString(errc));
		
		
		
		i++;
	}
	
	printf("Number of iterations: %d\nFinal epsilon: %.12lf\n", i, err);
	free(residual);
	free(residual_prev);
	free(search_direction);
	free(intermediate_comp);
    hipFree(d_A);
    hipFree(d_sd);
    hipFree(d_ic);
	hipFree(d_b);
    hipFree(d_res);
    hipFree(d_resp);
	hipFree(d_sol);
	return solution;
}	
