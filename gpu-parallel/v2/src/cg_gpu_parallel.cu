#include "hip/hip_runtime.h"
#include"cg_gpu_parallel.h"
#include"LAS_gpu_parallel.h"
#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define CACHE_BLOCK_SIZE 64 // bytes
#define BLOCK_DIM 32
// compute step size


// check where to use 
float* cg(unsigned int size, float* A, float* b, float* init_g, float epsilon, int debug)
{
	float*  solution;
	float*  residual;
	float*  residual_prev;
	float*  search_direction;
	float*  intermediate_comp;
	float alpha;
	float beta;
	float num;
	float denum;
	float err;
	int i = 0;
    printf("Size : %d \n", size);
	printf("Grid: %d , Block: %d\n", (size + BLOCK_DIM - 1) / BLOCK_DIM, BLOCK_DIM);	
	dim3 GridDim( (size + BLOCK_DIM - 1) / BLOCK_DIM ,1,1);
	dim3 BlockDim(BLOCK_DIM,1,1);

	// allocate host data
	solution = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	residual = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	residual_prev = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	search_direction = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);
	intermediate_comp = (float*)aligned_alloc(CACHE_BLOCK_SIZE, sizeof(float)*size);

	// allocate device data
	float* d_A, *d_b, *d_sd, *d_ic, *d_sol, *d_res, *d_resp;
	hipMalloc((void**)&d_A, sizeof(float)*size*size);
	hipMalloc((void**)&d_b, sizeof(float)*size);
	hipMalloc((void**)&d_sd, sizeof(float)*size);
	hipMalloc((void**)&d_ic, sizeof(float)*size);
	hipMalloc((void**)&d_sol, sizeof(float)*size);
	hipMalloc((void**)&d_res, sizeof(float)*size);
	hipMalloc((void**)&d_resp, sizeof(float)*size);


	// initialise host data
	scalar_vector_mult_inplace(size,solution,0);
	scalar_vector_mult_inplace(size,residual,0);
	scalar_vector_mult_inplace(size,residual_prev,0);
	scalar_vector_mult_inplace(size,search_direction,0);
	scalar_vector_mult_inplace(size,intermediate_comp,0);

	vector_copy(size, solution, init_g);
	
	// initialise device data
	hipMemcpy(d_A, A, sizeof(float)*size*size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(float)*size, hipMemcpyHostToDevice);
	hipMemcpy(d_sd, solution, sizeof(float)*size, hipMemcpyHostToDevice);
	hipMemcpy(d_ic, intermediate_comp, sizeof(float)*size, hipMemcpyHostToDevice);
	hipMemcpy(d_res, residual, sizeof(float)*size, hipMemcpyHostToDevice);
	hipMemcpy(d_resp, residual_prev, sizeof(float)*size, hipMemcpyHostToDevice);

	// residual calc
	// A = d_A , d_sd = solution , d_ic = residual
	matrix_vector_mult<<<GridDim,BlockDim>>>(size,d_A,d_sd,d_res);
	hipDeviceSynchronize();
	vector_sub<<<GridDim,BlockDim>>>(size,d_res,d_b,1.0,d_res);
	hipDeviceSynchronize();
	vector_copy<<<GridDim,BlockDim>>>(size,d_sd,d_res);
	hipDeviceSynchronize();
	scalar_vector_mult_inplace<<<GridDim,BlockDim>>>(size,d_sd,-1);
	hipDeviceSynchronize();

	hipMemcpy(residual,d_res,sizeof(float)*size,hipMemcpyDeviceToHost);

	err = dot_product(size, residual, residual);

	err = sqrt(err);

	

	while (err > epsilon)
	{
		
		printf("Launching kernel with GridDim=%d, BlockDim=%d\n", GridDim, BlockDim);

		test_access<<<GridDim,BlockDim>>>(d_sd);

		hipDeviceSynchronize();

		hipError_t errs = hipPeekAtLastError();
		if (errs != hipSuccess)
    			printf("Pre-launch error: %s\n", hipGetErrorString(errs));		

        matrix_vector_mult<<<GridDim,BlockDim>>>(size,d_A,d_sd,d_ic);
		
		hipDeviceSynchronize();
		hipError_t errc = hipGetLastError();
		if (errc != hipSuccess)
			printf("CUDA kernel error: %s\n", hipGetErrorString(errc));
        	hipMemcpy(intermediate_comp, d_ic, sizeof(float)*size,hipMemcpyDeviceToHost);
		
		hipMemcpy(residual,d_res,sizeof(float)*size,hipMemcpyDeviceToHost);
		hipMemcpy(intermediate_comp,d_ic,sizeof(float)*size,hipMemcpyDeviceToHost);
		num = dot_product(size, residual, residual);
		denum = dot_product(size, search_direction, intermediate_comp);
		
		alpha =  num / denum;
	
		vector_add<<<GridDim,BlockDim>>>(size,d_sol,d_sd,alpha,d_sol);
		hipDeviceSynchronize();
		//vector_copy(size,residual_prev,residual);
		
		vector_add<<<GridDim,BlockDim>>>(size,d_res,d_ic,alpha,d_res);
		hipDeviceSynchronize();
		denum = num;
		
		hipMemcpy(residual,d_res,sizeof(float)*size,hipMemcpyDeviceToHost);
		num = dot_product(size, residual,residual);

		//denum = dot_product(size, residual_prev, residual_prev);

		beta = num / denum;

		err = sqrt(num);
		
		vector_copy<<<GridDim,BlockDim>>>(size,d_ic,d_res);
		
		scalar_vector_mult_inplace<<<GridDim,BlockDim>>>(size,d_ic,-1);
		
		vector_add<<<GridDim,BlockDim>>>(size,d_ic,d_sd,beta,d_sd);

		i++;
	}
	
	printf("Number of iterations: %d\nFinal epsilon: %.12lf\n", i, err);
	free(residual);
	free(residual_prev);
	free(search_direction);
	free(intermediate_comp);
    hipFree(d_A);
    hipFree(d_sd);
    hipFree(d_ic);
	hipFree(d_b);
    hipFree(d_res);
    hipFree(d_resp);
	hipFree(d_sol);
	return solution;
}	
