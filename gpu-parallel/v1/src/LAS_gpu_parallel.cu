#include "hip/hip_runtime.h"
#include"LAS_gpu_parallel.h"
#include<hip/hip_runtime.h>

#include<stdio.h>
#include<stdlib.h>

// assume 1d Grid and 1d Block
__global__ void matrix_vector_mult(unsigned int size, float* matrix, float* vector, float* vector_storage)
{
<<<<<<< HEAD
  __global__ void matrix_vector_mult(unsigned int size, float* matrix, float* vector, float* vector_storage)
  {
      int row = blockDim.x*blockIdx.x + threadIdx.x;
      //int col = blockDim.y*blockIdx.y + threadIdx.y

      if (row < size)
      {
        float acc = 0;
        for (int i = 0; i < size; ++i)
        {
                acc += matrix[row*size+i]*vector[i];
        }
        vector_storage[row] = acc;
        //printf("vector_storage[%d] = %f\n", row, acc);
      }

      return;
  }
=======
	int row = blockDim.x*blockIdx.x + threadIdx.x;
  	//int col = blockDim.y*blockIdx.y + threadIdx.y

    	if (row < size)
	{
		float acc = 0;
		for (int i = 0; i < size; ++i)
		{
			acc += matrix[row*size+i]*vector[i];
		}
		vector_storage[row] = acc;
		//printf("vector_storage[%d] = %f\n", row, acc);
	}
	
	return;
>>>>>>> 15caaa05589a4119ffefa63a60776a11b54d0167
}

__global__ void test_access(float* vec) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx == 0) printf("vec[0] = %f\n", vec[0]);
}

float dot_product(unsigned int size, float* vector_1, float* vector_2)
{
	float storage = 0;
	for (unsigned int i = 0; i < size; ++i)
	{
		storage += vector_1[i]*vector_2[i]; 
	}
	return storage;
}

void vector_add(unsigned int size, float* vector_1, float* vector_2, float alpha, float* vector_storage)
{
	for (unsigned int i = 0; i < size; ++i)
	{
		vector_storage[i] = vector_1[i] + alpha*vector_2[i];	
	}
	return;
}

void vector_sub(unsigned int size, float* vector_1, float* vector_2, float alpha, float* vector_storage)
{
        for (unsigned int i = 0; i < size; ++i)
        {
                vector_storage[i] = vector_1[i] - alpha*vector_2[i];
        }
        return;
}

void vector_copy(unsigned int size, float* vector_out, float* vector_in)
{
	for (unsigned int i = 0; i < size; ++i)
        {
                vector_out[i] = vector_in[i];
        }
	return;
}

void scalar_vector_mult_inplace(unsigned int size,float* vector, float alpha)
{
	for (unsigned int i = 0; i < size; ++i)
	{
			vector[i] = alpha*vector[i];
	}
    return;
}

void print_vector(unsigned int size, float* vector)
{
	for (unsigned int i = 0; i < size; ++i)
	{
			printf("%lf ",vector[i]);
	}
	printf("\n");
}


/*
// assume 1d Grid and 1d Block
__global__ void dot_product(unsigned int size, float* vector_1, float* vector_2, float* vector_out)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < size)
  {
	  vector_out[i] = vector_1[i]*vector_2[i];
  }
  __syncthreads();
}

__global__ void vector_add(unsigned int size, float* vector_1, float* vector_2, float alpha, float* vector_storage)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < size)
  {
	  vector_out[i] = vector_1[i]+vector_2[i];
  }
  __syncthreads();
}

__global__ void vector_sub(unsigned int size, float* vector_1, float* vector_2, float alpha, float* vector_storage)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < size)
  {
	  vector_out[i] = vector_1[i]-vector_2[i];
  }
  __syncthreads();
}

__global__ void vector_copy(unsigned int size, float* vector_out, float* vector_in)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < size)
  {
	  vector_out[i] = vector_in[i];
  }
  __syncthreads();
}

__global__ void scalar_vector_mult_inplace(unsigned int size,float* vector, float alpha)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id < size)
  {
	  vector[i] = alpha*vector[i];
  }
  __syncthreads();
}
*/
